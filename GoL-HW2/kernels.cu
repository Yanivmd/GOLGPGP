#include "hip/hip_runtime.h"

#include "inc.h"


// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

/*
__global__ void packer(byte *d_in, int *d_packedIn, int sizeX, int sizeY)
{
	__shared__ int packedValues[NUM_THREADS_X*NUM_THREADS_Y/32];

	int myInIndex = (blockIdx.y*blockDim.y+threadIdx.y+1)*(sizeX+2) + (blockIdx.x*blockDim.x+threadIdx.x+1);
	int myOutIndex = ((blockIdx.y*blockDim.y+threadIdx.y)*(sizeX) + (blockIdx.x*blockDim.x+threadIdx.x)) / 32;
	if (myInIndex < ((sizeX+2)*(sizeY+2)))
	{
		int mySharedIndex = ((threadIdx.y * NUM_THREADS_X) + threadIdx.x)/32;
		packedValues[mySharedIndex] =0;
		int n1 = (int)(d_in[myInIndex]);
		n1 = n1 << (myInIndex % 32);

		atomicOr(&(packedValues[mySharedIndex]),n1);
		__syncthreads();

		d_packedIn[myOutIndex] = packedValues[mySharedIndex];
	}
}
*/
/*
//int packing
__global__ void packer(byte *d_in, int *d_packedIn, int sizeX, int sizeY)
{
	int myOutIndex = ((blockIdx.y*blockDim.y+threadIdx.y)*(sizeX) + (blockIdx.x*blockDim.x+threadIdx.x)) / 32;
	int myinIndex = (blockIdx.y*blockDim.y+threadIdx.y+1)*(sizeX+2) + (blockIdx.x*blockDim.x+threadIdx.x+1) ;
	if (myinIndex < ((sizeX+2)*(sizeY+2)))
	{
		int n1 = (((int)d_in[myinIndex]) << (myinIndex % 32)); 
		atomicOr(&(d_packedIn[myOutIndex]),n1);
	}
}
__global__ void unpacker(byte *d_out, int *d_packedOut, int sizeX, int sizeY)
{
	int myinIndex = ((blockIdx.y*blockDim.y+threadIdx.y)*(sizeX) + (blockIdx.x*blockDim.x+threadIdx.x)) / 32;
	int myOutIndex = (blockIdx.y*blockDim.y+threadIdx.y+1)*(sizeX+2) + (blockIdx.x*blockDim.x+threadIdx.x+1) ;
	if (myOutIndex < ((sizeX+2)*(sizeY+2)))
	{
		byte n1 = (d_packedOut[myinIndex] >> (myOutIndex % 32)) & 0x1; 
		d_out[myOutIndex] = n1;
	}
}
*/

// byte packing
/*__global__*/ void packer(byte *d_in, byte *d_packedIn, int sizeX, int sizeY, int bx, int by, int tx, int ty)
{
	//int row = blockIdx.y*blockDim.y+threadIdx.y;
	//int col = blockIdx.x*blockDim.x+threadIdx.x;
	int row = by*((sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y)+ty;
	int col = bx*((sizeX+NUM_THREADS_X-1)/NUM_THREADS_X)+tx;
	int outIndex = (row*sizeX+col);
	if (outIndex < (sizeX*sizeY)) {
		byte n1 = 0;
		for (int i=0; i<8; i++) {
			int inIndexMargin = (row+1)*(sizeX+2) + col + 1;
			n1 += ((d_in[inIndexMargin]) << (outIndex%8));
			col++;
			if (col>=sizeX) {
				row++;
				col -= sizeX;
			}
		}
		d_packedIn[outIndex/8] = n1;
	}
}
/*__global__*/ void unpacker(byte *d_out, byte *d_packedOut, int sizeX, int sizeY, int bx, int by, int tx, int ty)
{
	//int row = blockIdx.y*blockDim.y+threadIdx.y;
	//int col = blockIdx.x*blockDim.x+threadIdx.x;
	int row = by*((sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y)+ty;
	int col = bx*((sizeX+NUM_THREADS_X-1)/NUM_THREADS_X)+tx;
	int outIndexMargin = (row+1)*(sizeX+2) + col + 1;
	int inIndex = (row*sizeX+col);
	if (inIndex < (sizeX*sizeY)) {
		byte n1 = (d_packedOut[inIndex/8] >> (inIndex%8)) & 0x1;
		d_out[outIndexMargin]=n1;
	}
}


int host(int sizeX, int sizeY, byte* input, byte* output, int iterations, string outfilename)
{
	byte *d_in=NULL, *d_out=NULL;
	int *d_packedIn=NULL, *d_packedOut=NULL;
	int *d_generations=NULL;

	int field_size = (sizeX+2)*(sizeY+2);

	int numBlocks = ((sizeX+NUM_THREADS_X-1)/NUM_THREADS_X) * ((sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y);

	checkCudaErrors(hipMalloc((void**)&d_in,field_size*sizeof(byte)));
	checkCudaErrors(hipMalloc((void**)&d_out,field_size*sizeof(byte)));
	checkCudaErrors(hipMalloc((void**)&d_packedIn,(field_size/8)));
	checkCudaErrors(hipMalloc((void**)&d_packedOut,(field_size/8)));
	checkCudaErrors(hipMalloc((void**)&d_generations,numBlocks*sizeof(int)));

	hipMemset(d_out, 0, field_size); //TODO delete
	hipMemset(d_packedIn, 0, (field_size/8)); 
	hipMemset(d_generations, 0, numBlocks*sizeof(int));

	checkCudaErrors(hipMemcpy(d_in, input, field_size, hipMemcpyHostToDevice));

	hipError_t err;

	byte* packed = new byte[sizeX*sizeY];
	for (int i=0; i<(sizeX+NUM_THREADS_X-1)/NUM_THREADS_X; i++)
		for (int j=0; j<(sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y; j++)
			for (int l=0; l<NUM_THREADS_X; l++)
				for (int k=0; k<NUM_THREADS_Y; k++)
					packer(input,packed,sizeX,sizeY,i,j,l,k);

	for (int i=0; i<(sizeX+NUM_THREADS_X-1)/NUM_THREADS_X; i++)
		for (int j=0; j<(sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y; j++)
			for (int l=0; l<NUM_THREADS_X; l++)
				for (int k=0; k<NUM_THREADS_Y; k++)
					unpacker(output,packed,sizeX,sizeY,i,j,l,k);
	int res  = memcmp(input,output,field_size);
	free(packed);
	/*
	dim3 packerThreads(NUM_THREADS_X,NUM_THREADS_Y);
	dim3 packerGrid((sizeX+NUM_THREADS_X-1)/NUM_THREADS_X,(sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y);

	packer<<<packerGrid,packerThreads>>>(d_in, d_packedIn, sizeX, sizeY);

	if ((err = hipGetLastError()) != hipSuccess)
    {
    	printf("packer launch failed: %s",hipGetErrorString(err));
    	exit(1);
    }

	hipEvent_t start,stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, NULL));

	// Setup execution parameters
	dim3 threads(NUM_THREADS_X,NUM_THREADS_Y);
	dim3 grid(NUM_BLOCKS_X,NUM_BLOCKS_Y);
	//kernel<<<grid,threads>>>(d_in, d_out, sizeX, sizeY, iterations, d_generations);

	if ((err = hipGetLastError()) != hipSuccess)
    {
    	printf("Kernel launch failed: %s",hipGetErrorString(err));
    	exit(1);
    }

	checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

	//TODO   d_packedOut
	unpacker<<<packerGrid,packerThreads>>>(d_out, d_packedIn, sizeX, sizeY);

    // check for errors during kernel launch
   
    if ((err = hipGetLastError()) != hipSuccess)
    {
    	printf("unpacker launch failed: %s",hipGetErrorString(err));
    	exit(1);
    }
	
    float msec = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msec, start, stop));
	

	//TODO!!!!! change back to 0
	if (iterations % 2 == 1) {
		hipMemcpy(output, d_in, field_size, hipMemcpyDeviceToHost);
		hipMemcpy(input, d_out, field_size, hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(output, d_out, field_size, hipMemcpyDeviceToHost);
		hipMemcpy(input, d_in, field_size, hipMemcpyDeviceToHost);
	}


//	int res  = memcmp(input,output,field_size);

//	printf("%dx%d field size, %d generation, %f ms\n",sizeX,sizeY,iterations,msec);

    hipFree(d_in);
    hipFree(d_out);
    //hipEventDestroy(start);
    //hipEventDestroy(stop);
	*/
	return 0;
}

__global__ void kernel(
		byte* d_in,
		byte* d_out,
		int sizeX,
		int sizeY,
		int iterations,
		int* blockGenerations
		)
{
	int sizeXmargin = sizeX + 2;

	int maxVirtualBlockY = (sizeY+NUM_THREADS_Y-1)/NUM_THREADS_Y;
	int maxVirtualBlockX = (sizeX+NUM_THREADS_X-1)/NUM_THREADS_X;

	int virtaulBlockPerSM = (maxVirtualBlockY*maxVirtualBlockX)/gridDim.x;
	int memoryPerVirtualBlock = (blockDim.x+2)*(blockDim.y+2);
//	__shared__ byte mem[memoryPerVirtualBlock*virtaulBlockPerSM];

	byte* in = d_in;
	byte* out = d_out;

	for (int k=0; k<iterations; k++) {
		int virtualBlockY = blockIdx.y; + (blockIdx.x / maxVirtualBlockX);
		int virtualBlockX = blockIdx.x % maxVirtualBlockX;
		while (virtualBlockY < maxVirtualBlockY) {
			while (virtualBlockX < maxVirtualBlockX) {
				int absRow = (virtualBlockY * NUM_THREADS_Y) + threadIdx.y;
				int absCol = (virtualBlockX * NUM_THREADS_X) + threadIdx.x;

				int absGenLocInArray = (virtualBlockY * maxVirtualBlockX) + virtualBlockX;
				__threadfence_system();
				for (int i=-1; i<=1; i++) {
					for (int j=-1; j<=1; j++) {
						int genIndex = (i * maxVirtualBlockX) + j + absGenLocInArray;
						if ((genIndex >= 0) && (genIndex < (maxVirtualBlockX * maxVirtualBlockY)))
							while (blockGenerations[genIndex] < k)
								__threadfence_system();
					}
				}

				if ((absRow < sizeY) && (absCol < sizeX)) {
					int absLocInArray = ((absRow+1) * sizeXmargin) + (absCol+1);
				

					byte* ptr = &in[absLocInArray];
					//TODO check neighbors vector
					int neighbors = 0;

					neighbors += ptr[-1 * sizeXmargin + -1];
					neighbors += ptr[-1 * sizeXmargin +  0];
					neighbors += ptr[-1 * sizeXmargin +  1];
					neighbors += ptr[ 0 * sizeXmargin + -1];
					neighbors += ptr[ 0 * sizeXmargin +  1];
					neighbors += ptr[ 1 * sizeXmargin + -1];
					neighbors += ptr[ 1 * sizeXmargin +  0];
					neighbors += ptr[ 1 * sizeXmargin +  1];

					if (neighbors == 3 ||
						(ptr[0] == ALIVE && neighbors == 2) ) {
						out[absLocInArray] = ALIVE;
					}
					else {
						out[absLocInArray] = DEAD;
					}
				}
							
				__syncthreads();
			
				// this is not necessary on last iteration
				blockGenerations[absGenLocInArray] = k+1;

				virtualBlockX += gridDim.x;

			}
			virtualBlockY += virtualBlockX / maxVirtualBlockX;
			virtualBlockX = virtualBlockX % maxVirtualBlockX;
		}

		byte* tmp = in;
		in=out;
		out=tmp;
	}
}