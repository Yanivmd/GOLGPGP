#include "hip/hip_runtime.h"
#include "inc.h"

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
	if(hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

void fillMargin(int* field, int sizeX, int sizeY,int val)
{
	for (int r = 0; r < sizeY; r++)
	{
		field[r * sizeX] = val;
		field[r * sizeX + sizeX-1] = val;
	}
	std::fill_n(&field[0],sizeX,val);
	std::fill_n(&field[sizeX*(sizeY-1)],sizeX,val);
}

byte* host(int numberOfCols, int numberOfRows, byte* input, int iterations)
{
	if (iterations == 0) {
		return input;
	}

	byte* output = NULL;

	// globals on CPU
	int *blockGenerations;
	byte *bordersArray;

	// Globals on GPU
	byte *d_in=NULL, *d_out=NULL;
	int *d_blockGenerations=NULL;
	byte *d_bordersArray = NULL;
	byte *d_bordersArray2 = NULL;

	const int numberOfVirtualBlockY = (numberOfRows+NUM_THREADS_Y-1)/NUM_THREADS_Y;
	const int numberOfVirtualBlockX = (numberOfCols+NUM_THREADS_X-1)/NUM_THREADS_X;

	int numOfBlocks = (numberOfVirtualBlockY+GEN_MARGIN_SIZE)*(numberOfVirtualBlockX+GEN_MARGIN_SIZE);

	checkCudaErrors(hipMalloc((void**)&d_blockGenerations,numOfBlocks*sizeof(int)));
	blockGenerations = new int[numOfBlocks];
	std::fill_n(blockGenerations,numOfBlocks,1);
	fillMargin(blockGenerations,numberOfVirtualBlockX+GEN_MARGIN_SIZE,numberOfVirtualBlockY+GEN_MARGIN_SIZE,iterations);

	int sizeOfBordersAry = (numberOfVirtualBlockY+GEN_MARGIN_SIZE)*(numberOfVirtualBlockX+GEN_MARGIN_SIZE)*(NUM_THREADS_X*2+NUM_THREADS_Y*2);

	checkCudaErrors(hipMalloc((void**)&d_bordersArray,sizeOfBordersAry*sizeof(byte)));
	checkCudaErrors(hipMalloc((void**)&d_bordersArray2,sizeOfBordersAry*sizeof(byte)));
	bordersArray = new byte[sizeOfBordersAry];
	std::fill_n(bordersArray,sizeOfBordersAry,0);

	checkCudaErrors(hipMemcpy(d_blockGenerations, blockGenerations, numOfBlocks*sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_bordersArray, bordersArray, sizeOfBordersAry*sizeof(byte), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_bordersArray2, bordersArray, sizeOfBordersAry*sizeof(byte), hipMemcpyHostToDevice));

	// original stuff
	int field_size = (numberOfCols+GLOBAL_MARGIN_SIZE)*(numberOfRows+GLOBAL_MARGIN_SIZE);
	checkCudaErrors(hipMalloc((void**)&d_in,field_size*sizeof(byte)));
	checkCudaErrors(hipMalloc((void**)&d_out,field_size*sizeof(byte)));

	hipMemset(d_out, 0, field_size); //TODO delete
	checkCudaErrors(hipMemcpy(d_in, input, field_size, hipMemcpyHostToDevice));

	dim3 threads(NUM_THREADS_X,NUM_THREADS_Y);
	dim3 grid(NUM_BLOCKS_X,1);

#ifdef MEASUREMENTS
	hipEvent_t start,stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, NULL));

	int iters=10;
	for (int i=0; i<iters; i++) {
#endif
		kernel<<<grid,threads>>>(d_in,d_out,numberOfRows,numberOfCols,numberOfVirtualBlockX,numberOfVirtualBlockY,iterations,d_bordersArray,d_bordersArray2,d_blockGenerations);
#ifdef MEASUREMENTS
	}

	checkCudaErrors(hipEventRecord(stop, NULL));
	checkCudaErrors(hipEventSynchronize(stop));

	float msec = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msec, start, stop));
	msec /= iters;

	printf("%dx%d field size, %d generation, %f ms\n",numberOfCols,numberOfRows,iterations,msec);
#endif

	output = new byte[field_size*sizeof(byte)];
	hipMemcpy(output, d_out, field_size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_blockGenerations);
	hipFree(d_bordersArray);

	delete[] bordersArray;
	delete[] blockGenerations;

	return output;
}

const int gridDimx = NUM_BLOCKS_X;

const int blockDimx = NUM_THREADS_X;
const int blockDimy = NUM_THREADS_Y;

const int totalNumberOfVbsY = ((MAX_NUMBER_ROWS+NUM_THREADS_Y-1)/NUM_THREADS_Y);  //rows
const int totalNumberOfVbsX = ((MAX_NUMBER_COLS+NUM_THREADS_X-1)/NUM_THREADS_X);  //cols

const int totalNumberOfVBs =  totalNumberOfVbsY*totalNumberOfVbsX ;
const int totalVirtaulBlocksPerSM = (totalNumberOfVBs) / gridDimx;


// TODO - all y!=0 are waisted...
__forceinline__ __device__ void share2glob(byte * blockWithMargin,byte *BordersAryPlace,int usedColsNoMar, int usedRowsNoMar, int totalCols,int totalRows, byte numberOfWarpsToUse)
{

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int totalColsWithMar = totalCols+ MARGIN_SIZE_COLS;

	byte *row2Fill;
	int writeIndex;

	int dev8 = tx;

	if (ty % numberOfWarpsToUse == (0 % numberOfWarpsToUse))
	{

		// copy border UP
		row2Fill = getUPBorder(BordersAryPlace,totalCols,totalRows);
		writeIndex = dev8;
		for (int row=1;row<=1;row++)
		{
			for (int col=1+dev8;col<=usedColsNoMar;col+=32)
			{
				row2Fill[writeIndex] = blockWithMargin[row * (totalColsWithMar) + col];
				writeIndex +=32;
			}
		}
	}

	if (ty % numberOfWarpsToUse == (1 % numberOfWarpsToUse))
	{
		// copy border Down
		row2Fill = getDOWNBorder(BordersAryPlace,totalCols,totalRows);
		writeIndex = dev8;
		for (int row=1+usedRowsNoMar-1;row<=1+usedRowsNoMar-1;row++)
		{
			for (int col=1+dev8;col<=usedColsNoMar;col+=32)
			{	
				row2Fill[writeIndex] = blockWithMargin[row * (totalColsWithMar) + col];
				writeIndex +=32;
			}
		}
	}

	if (ty % numberOfWarpsToUse == (2 % numberOfWarpsToUse))
	{
		// copy border LEFT
		row2Fill = getLEFTBorder(BordersAryPlace,totalCols,totalRows);
		writeIndex = dev8;
		for (int row=1 +dev8;row<=usedRowsNoMar;row+=32)
		{	
			for (int col=1;col<=1;col++)
			{
				// move past margin, then skip n rows...
				row2Fill[writeIndex] = blockWithMargin[row * (totalColsWithMar) + col];
				writeIndex +=32;
			}
		}
	}

	if (ty % numberOfWarpsToUse == (3 % numberOfWarpsToUse))
	{
		// copy border Right
		row2Fill = getRIGHTBorder(BordersAryPlace,totalCols,totalRows);
		writeIndex = dev8;
		for (int row=1 +dev8;row<=usedRowsNoMar;row+=32)
		{	
			for (int col=usedColsNoMar;col<=usedColsNoMar;col++)
			{
				// move past margin, then skip n rows...
				row2Fill[writeIndex] = blockWithMargin[row * (totalColsWithMar) + col];
				writeIndex +=32;
			}
		}
	}
}


__forceinline__ __device__ void fillBorders(byte * blockWithMargin,byte *fullBordersArry,int VBx,int VBy,int totalVBCols,
		int usedColsNoMar, int usedRowsNoMar, int totalCols,int totalRows, byte numberOfWarpsToUse)
{

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	// ajust to margin 
	//VBx +=1;
	//VBy +=1;

	const int totalColsWithMar = totalCols+ MARGIN_SIZE_COLS;
	const int totalRowsWithMar = totalRows + MARGIN_SIZE_ROWS;
	byte* borderPtr;

	if (ty % numberOfWarpsToUse == (0 % numberOfWarpsToUse))
	{
		// LEFT UP
		borderPtr = getDOWNBorder(getBordersVBfromXY(fullBordersArry,VBx-1,VBy-1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		blockWithMargin[0*totalColsWithMar+0] = borderPtr[totalCols-1]; // -1 , cuz 0 based. (no margin!!!)

		// UP
		borderPtr = getDOWNBorder(getBordersVBfromXY(fullBordersArry,VBx,VBy-1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		for (int col=1+tx;col<totalColsWithMar-(MARGIN_SIZE_COLS-2)-1;col+=32)
		{
			blockWithMargin[0*totalColsWithMar+col] = borderPtr[col-1];
		}
	}

	if (ty % numberOfWarpsToUse ==(1 % numberOfWarpsToUse))
	{
		// RIGHT UP
		borderPtr = getDOWNBorder(getBordersVBfromXY(fullBordersArry,VBx+1,VBy-1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		blockWithMargin[0*totalColsWithMar + (usedColsNoMar+1)] = borderPtr[0]; 

		// LEFT
		byte * ptr1 = getBordersVBfromXY(fullBordersArry,VBx-1,VBy,totalVBCols,totalCols,totalRows);
		borderPtr = getRIGHTBorder(ptr1,totalCols,totalRows);
		for (int row=1+tx;row<totalRowsWithMar-(MARGIN_SIZE_ROWS-2)-1 ;row+=32)
		{
			blockWithMargin[row*totalColsWithMar + 0] = borderPtr[row-1];
		}
	}

	if (ty % numberOfWarpsToUse ==(2 % numberOfWarpsToUse))
	{
		// RIGHT
		byte * ptr2 = getBordersVBfromXY(fullBordersArry,VBx+1,VBy,totalVBCols,totalCols,totalRows);
		borderPtr = getLEFTBorder(ptr2,totalCols,totalRows);
		for (int row=1+tx;row<totalRowsWithMar-(MARGIN_SIZE_ROWS-2) -1 ;row+=32)
		{
			blockWithMargin[row*totalColsWithMar + (usedColsNoMar+1)] = borderPtr[row-1];
		}

		// DOWN LEFT
		borderPtr = getUPBorder(getBordersVBfromXY(fullBordersArry,VBx-1,VBy+1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		blockWithMargin[(usedRowsNoMar +1) * totalColsWithMar + 0] = borderPtr[totalCols-1]; // -1 cuz 0 based  . (no margin!!!)
	}
	if (ty % numberOfWarpsToUse ==(3 % numberOfWarpsToUse))
	{
		// DOWN
		borderPtr = getUPBorder(getBordersVBfromXY(fullBordersArry,VBx,VBy+1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		for (int col=1+tx;col<=totalColsWithMar-MARGIN_SIZE_COLS;col+=32)
		{
			blockWithMargin[(usedRowsNoMar +1)*totalColsWithMar+col] = borderPtr[col-1];
		}

		// DOWN RIGHT
		borderPtr = getUPBorder(getBordersVBfromXY(fullBordersArry,VBx+1,VBy+1,totalVBCols,totalCols,totalRows),totalCols,totalRows);
		blockWithMargin[(usedRowsNoMar +1) * totalColsWithMar + (usedColsNoMar+1)] = borderPtr[0]; 
	}
}







__forceinline__ __device__  void packer(byte* in, byte* out, int numUsedCols, int numUsedRows, int numTotalCols, int numTotalRows)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	int roundedTotalCols = (numTotalCols+7)/8;
	int col = tx%roundedTotalCols;
	int row = ty*8 + (tx/roundedTotalCols);
	int outIndex = row*roundedTotalCols+col;
	int inIndexMargin = (row+1)*(numTotalCols+MARGIN_SIZE_COLS) + col*8 + 1;
	if ((row < numUsedRows) && (col < numUsedCols)) {
		byte n1 = 0;
		for (int i=0; i<8 && (col < numUsedCols); i++) {
			n1 |= in[inIndexMargin] << (i%8);
			col++;
			inIndexMargin++;
		}
		out[outIndex] = n1;
	}
}

__forceinline__ __device__  void unpacker(byte* in, byte* out, int numUsedCols, int numUsedRows, int numTotalCols, int numTotalRows)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	int roundedTotalCols = (numTotalCols+7)/8;
	int inIndex = ty*roundedTotalCols+tx/8;
	int outIndexMargin = (ty+1)*(numTotalCols+MARGIN_SIZE_COLS) + tx + 1;
	if ((tx < numUsedCols) && (ty < numUsedRows)) {
		byte n1 = (in[inIndex] >> (tx%8)) & 0x1;
		out[outIndexMargin] = n1;
	}
} 

__forceinline__ __device__ void check(int numberOfVirtualBlockX,int numberOfVirtualBlockY, int absGenLocInArray,int * blockGenerations,int k)
{
	__threadfence_system();
#pragma unroll
	for (int i=-1; i<=1; i++) {
#pragma unroll
		for (int j=-1; j<=1; j++) {
			int genIndex = (i * numberOfVirtualBlockX) + j + absGenLocInArray;
			if ((genIndex >= 0) && (genIndex < (numberOfVirtualBlockX * numberOfVirtualBlockY)))
			{
				while (blockGenerations[genIndex] < k)
					__threadfence_system();
			}
		}
	}
}

// tx - 0,31 ; ty=0,29
__forceinline__ __device__ void  eval(byte * srcBlockWithMargin,byte * tarBlockWithMargin,int totalCols, int totalRows)
{
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	// i assume the check done to see if we can cals 
	int numberOfColsWithMar = totalCols + MARGIN_SIZE_COLS;
	byte *ptr = &(srcBlockWithMargin[((ty+1) * numberOfColsWithMar) + (tx+1)]);
	byte *out = &(tarBlockWithMargin[((ty+1) * numberOfColsWithMar) + (tx+1)]);
	//TODO check neighbors vector
	int neighbors = 0;

	neighbors += ptr[-1 * numberOfColsWithMar + -1];
	neighbors += ptr[-1 * numberOfColsWithMar +  0];
	neighbors += ptr[-1 * numberOfColsWithMar +  1];
	neighbors += ptr[ 0 * numberOfColsWithMar + -1];
	neighbors += ptr[ 0 * numberOfColsWithMar +  1];
	neighbors += ptr[ 1 * numberOfColsWithMar + -1];
	neighbors += ptr[ 1 * numberOfColsWithMar +  0];
	neighbors += ptr[ 1 * numberOfColsWithMar +  1];

	if (neighbors == 3 ||
			(ptr[0] == ALIVE && neighbors == 2) ) {
		*out = ALIVE;
	}
	else {
		*out = DEAD;
	}	
}

__global__ void kernel(byte* input, byte* output,const int numberOfRows,const int numberOfCols,
		int numberOfVirtualBlockX, int numberOfVirtualBlockY,
		int iterations, byte *bordersArray, byte *bordersArray2, int * blockGenerations)
{
	const int memoryPerVirtualBlock = (blockDimx+MARGIN_SIZE_COLS)*(blockDimy+MARGIN_SIZE_ROWS);

	__shared__ byte work__shared__[memoryPerVirtualBlock];
	__shared__ byte work2__shared__[memoryPerVirtualBlock];

	// TODO - we only really need to zero the margin...
	for (int i=threadIdx.x;i<memoryPerVirtualBlock;i+=blockDim.x)
	{
		work__shared__[i] = 0;
		work2__shared__[i] = 0;	
	}

	const int sizeOfPackedVB = ((blockDimx+7)/8)*blockDimy; 
	__shared__ byte packed__shared__[sizeOfPackedVB*totalVirtaulBlocksPerSM];

	byte* bordersIn = bordersArray;
	byte* bordersOut = bordersArray2;

	byte *currentWork;
	byte *nextWork;

	currentWork = work__shared__;
	nextWork =  work2__shared__;

	byte* in = input; //was d_
	byte* out = output; // was d_

	// DOR 0 - read from global
	int virtualGlobalBlockY = blockIdx.y + (blockIdx.x / numberOfVirtualBlockX);
	int virtualGlobalBlockX = blockIdx.x % numberOfVirtualBlockX;

	int packedIndex = 0;

	while (virtualGlobalBlockY < numberOfVirtualBlockY) {
		while (virtualGlobalBlockX < numberOfVirtualBlockX) {

			int usedCols = min(NUM_THREADS_X,numberOfCols-(virtualGlobalBlockX * NUM_THREADS_X));
			int usedRows = min(NUM_THREADS_Y,numberOfRows-(virtualGlobalBlockY * NUM_THREADS_Y));

			int absRow = (virtualGlobalBlockY * NUM_THREADS_Y) + threadIdx.y;
			int absCol = (virtualGlobalBlockX * NUM_THREADS_X) + threadIdx.x;

			if ((absRow < numberOfRows) && (absCol < numberOfCols)) {
				int numberOfColsWithMar = numberOfCols+GLOBAL_MARGIN_SIZE;
				byte *ptr = &(in[(absRow+1)*(numberOfCols+GLOBAL_MARGIN_SIZE)+absCol+1]);
				byte *out = &(nextWork[(threadIdx.y+1)*(NUM_THREADS_X+MARGIN_SIZE_COLS)+threadIdx.x+1]);
				int neighbors = 0;

				neighbors += ptr[-1 * numberOfColsWithMar + -1];
				neighbors += ptr[-1 * numberOfColsWithMar +  0];
				neighbors += ptr[-1 * numberOfColsWithMar +  1];
				neighbors += ptr[ 0 * numberOfColsWithMar + -1];
				neighbors += ptr[ 0 * numberOfColsWithMar +  1];
				neighbors += ptr[ 1 * numberOfColsWithMar + -1];
				neighbors += ptr[ 1 * numberOfColsWithMar +  0];
				neighbors += ptr[ 1 * numberOfColsWithMar +  1];

				if (neighbors == 3 ||
						(ptr[0] == ALIVE && neighbors == 2) ) {
					*out = ALIVE;
				}
				else {
					*out = DEAD;
				}
			}

			__syncthreads();

			if (iterations==1) {
				if ((absRow < numberOfRows) && (absCol < numberOfCols)) {
					out[(absRow+1)*(numberOfCols+GLOBAL_MARGIN_SIZE)+absCol+1] = nextWork[(threadIdx.y+1)*(NUM_THREADS_X+MARGIN_SIZE_COLS)+threadIdx.x+1];
				}
			} else {
				if (threadIdx.y < usedRows) {
					if ((threadIdx.y < usedRows) && (threadIdx.y < WARPS_FOR_PACKING))
						packer(nextWork,&packed__shared__[packedIndex*sizeOfPackedVB],usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y);
				}

				if ((WARPS_FOR_PACKING <= threadIdx.y) && (threadIdx.y < (WARPS_FOR_PACKING + WARPS_FOR_BORDERS))) {
					share2glob(nextWork,getBordersVBfromXY(bordersIn,virtualGlobalBlockX,virtualGlobalBlockY,numberOfVirtualBlockX,NUM_THREADS_X,NUM_THREADS_Y),
							usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y,WARPS_FOR_BORDERS);
				}
			}

			// TODO - check if necessary
			__syncthreads();

			virtualGlobalBlockX += gridDimx;
			packedIndex +=1;

			byte* tmp = nextWork;
			nextWork=currentWork;
			currentWork=tmp;
		}

		virtualGlobalBlockY += virtualGlobalBlockX / numberOfVirtualBlockX;
		virtualGlobalBlockX = virtualGlobalBlockX % numberOfVirtualBlockX;
	}

	if (iterations == 1) {
		return;
	}

	__syncthreads(); // tODO check if necessary

	// this was once for k= iterations...
	for (int k=1; k<iterations-1; k++)
	{

		int virtualGlobalBlockY = blockIdx.y + (blockIdx.x / numberOfVirtualBlockX);
		int virtualGlobalBlockX = blockIdx.x % numberOfVirtualBlockX;

		int packedIndex = 0;

		while (virtualGlobalBlockY < numberOfVirtualBlockY) {
			while (virtualGlobalBlockX < numberOfVirtualBlockX) {

				int absGenLocInArray = (virtualGlobalBlockY * numberOfVirtualBlockX) + virtualGlobalBlockX;
				int usedCols = min(NUM_THREADS_X,numberOfCols-(virtualGlobalBlockX * NUM_THREADS_X));
				int usedRows = min(NUM_THREADS_Y,numberOfRows-(virtualGlobalBlockY * NUM_THREADS_Y));

				check(numberOfVirtualBlockX,numberOfVirtualBlockY,absGenLocInArray,blockGenerations,k);

				if (((WARPS_FOR_PACKING + WARPS_FOR_BORDERS) <= threadIdx.y) && (threadIdx.y < (WARPS_FOR_PACKING + WARPS_FOR_BORDERS + WARPS_FOR_BORDERS))) {
					fillBorders(currentWork,bordersIn,virtualGlobalBlockX,virtualGlobalBlockY,((numberOfCols+NUM_THREADS_X-1)/NUM_THREADS_X),usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y,WARPS_FOR_BORDERS);
				}

				unpacker(&packed__shared__[packedIndex*sizeOfPackedVB],currentWork,usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y);

				__syncthreads();

				int absRow = (virtualGlobalBlockY * NUM_THREADS_Y) + threadIdx.y;
				int absCol = (virtualGlobalBlockX * NUM_THREADS_X) + threadIdx.x;


				if ((absRow < numberOfRows) && (absCol < numberOfCols)) {
					eval(currentWork,nextWork,NUM_THREADS_X,NUM_THREADS_Y);
				}

				__syncthreads();

				if ((threadIdx.y < usedRows) && (threadIdx.y < WARPS_FOR_PACKING)) {
					packer(nextWork,&packed__shared__[packedIndex*sizeOfPackedVB],usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y);
				}

				if ((WARPS_FOR_PACKING <= threadIdx.y) && (threadIdx.y < (WARPS_FOR_PACKING + WARPS_FOR_BORDERS))) {
					share2glob(nextWork,getBordersVBfromXY(bordersOut,virtualGlobalBlockX,virtualGlobalBlockY,numberOfVirtualBlockX,NUM_THREADS_X,NUM_THREADS_Y),
							usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y,WARPS_FOR_BORDERS);
				}

				__syncthreads();

				// this is not necessary on last iteration
				// "NOTIFY"
				blockGenerations[absGenLocInArray] = k+1;

				virtualGlobalBlockX += gridDimx;
				packedIndex +=1;

				byte* tmp = nextWork;
				nextWork=currentWork;
				currentWork=tmp;
			}

			virtualGlobalBlockY += virtualGlobalBlockX / numberOfVirtualBlockX;
			virtualGlobalBlockX = virtualGlobalBlockX % numberOfVirtualBlockX;
		}

		byte* tmp = bordersIn;
		bordersIn = bordersOut;
		bordersOut = tmp;
	}

	// DOR K - write to global
	virtualGlobalBlockY = blockIdx.y + (blockIdx.x / numberOfVirtualBlockX);
	virtualGlobalBlockX = blockIdx.x % numberOfVirtualBlockX;

	packedIndex = 0;

	while (virtualGlobalBlockY < numberOfVirtualBlockY) {
		while (virtualGlobalBlockX < numberOfVirtualBlockX) {

			int usedCols = min(NUM_THREADS_X,numberOfCols-(virtualGlobalBlockX * NUM_THREADS_X));
			int usedRows = min(NUM_THREADS_Y,numberOfRows-(virtualGlobalBlockY * NUM_THREADS_Y));

			int absRow = (virtualGlobalBlockY * NUM_THREADS_Y) + threadIdx.y;
			int absCol = (virtualGlobalBlockX * NUM_THREADS_X) + threadIdx.x;

			if (((WARPS_FOR_PACKING + WARPS_FOR_BORDERS) <= threadIdx.y) && (threadIdx.y < (WARPS_FOR_PACKING + WARPS_FOR_BORDERS + WARPS_FOR_BORDERS))) {
				fillBorders(currentWork,bordersIn,virtualGlobalBlockX,virtualGlobalBlockY,((numberOfCols+NUM_THREADS_X-1)/NUM_THREADS_X),usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y,WARPS_FOR_BORDERS);
			}

			unpacker(&packed__shared__[packedIndex*sizeOfPackedVB],currentWork,usedCols,usedRows,NUM_THREADS_X,NUM_THREADS_Y);

			__syncthreads();

			if ((absRow < numberOfRows) && (absCol < numberOfCols)) {
				int numberOfColsWithMar = NUM_THREADS_X+MARGIN_SIZE_COLS;
				byte *ptr = &(currentWork[(threadIdx.y+1)*(NUM_THREADS_X+MARGIN_SIZE_COLS)+threadIdx.x+1]);
				byte *outPtr = &(out[(absRow+1)*(numberOfCols+GLOBAL_MARGIN_SIZE)+absCol+1]);
				int neighbors = 0;

				neighbors += ptr[-1 * numberOfColsWithMar + -1];
				neighbors += ptr[-1 * numberOfColsWithMar +  0];
				neighbors += ptr[-1 * numberOfColsWithMar +  1];
				neighbors += ptr[ 0 * numberOfColsWithMar + -1];
				neighbors += ptr[ 0 * numberOfColsWithMar +  1];
				neighbors += ptr[ 1 * numberOfColsWithMar + -1];
				neighbors += ptr[ 1 * numberOfColsWithMar +  0];
				neighbors += ptr[ 1 * numberOfColsWithMar +  1];

				if (neighbors == 3 ||
						(ptr[0] == ALIVE && neighbors == 2) ) {
					*outPtr = ALIVE;
				}
				else {
					*outPtr = DEAD;
				}
			}

			virtualGlobalBlockX += gridDimx;
			packedIndex +=1;

			byte* tmp = nextWork;
			nextWork=currentWork;
			currentWork=tmp;

		}

		virtualGlobalBlockY += virtualGlobalBlockX / numberOfVirtualBlockX;
		virtualGlobalBlockX = virtualGlobalBlockX % numberOfVirtualBlockX;
	}
}
